#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define NUM_THREADS 8
#define GROUP_SIZE 8
__global__ void mandelKernel(int *d_data,
                             int width,
                             float stepX, float stepY,
                             float lowerX, float lowerY,
                             int maxIteration,size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //

    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float c_im = lowerY + thisY * stepY;
    int i;
    int* row = (int*)((char*)d_data + thisY * pitch);
    
    for (int j = 0; j < GROUP_SIZE; j++) {
      float c_re = lowerX + thisX * stepX;
      float z_re = c_re, z_im = c_im;
      
      for (i = 0; i < maxIteration; i++)
      {
        if (z_re * z_re + z_im * z_im > 4.f)
          break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
      }
      
      row[thisX] = i;
      thisX++;
    }

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int size = resX * resY * sizeof(int);
    int *h_out;
    int *d_out;
    size_t pitch;
    hipHostAlloc((void **)&h_out, size, hipHostMallocDefault);
    hipMallocPitch((void **)&d_out, &pitch, sizeof(int)*resX, resY);
    dim3 block(NUM_THREADS, NUM_THREADS);
    dim3 grid(resX / (block.x*GROUP_SIZE), resY / block.y);
    
    mandelKernel<<<grid, block>>>(d_out, resX, stepX, stepY, lowerX, lowerY, maxIterations, pitch);

    hipMemcpy2D(h_out, resX*sizeof(int), d_out, pitch, resX*sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_out, size);
    
    hipHostFree(h_out);
    hipFree(d_out);
}
