#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define NUM_THREADS 8
__global__ void mandelKernel(int *d_data,
                             int width,
                             float stepX, float stepY,
                             float lowerX, float lowerY,
                             int maxIteration) {
    // To avoid error caused by the floating number, use the following pseudo code
    //

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIteration; ++i)
    {

      if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = c_re + new_re;
      z_im = c_im + new_im;
    }
    
    d_data[thisX + thisY * width] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int size = resX * resY * sizeof(int);
    int *h_out = (int *) malloc(size);
    int *d_out;
    hipMalloc(&d_out, size);

    dim3 block(NUM_THREADS, NUM_THREADS);
    dim3 grid(resX / NUM_THREADS, resY / NUM_THREADS);
    mandelKernel<<<grid, block>>>(d_out, resX, stepX, stepY, lowerX, lowerY, maxIterations);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    memcpy(img, h_out, size);
    hipFree(d_out);
    free(h_out);
}
